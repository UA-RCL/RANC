/// corecontroller.cpp
/// 
/// Created for the University of Arizona Reconfigurable Computing Lab
///
///

#include <iostream>
#include <sstream>
#include <bitset>
#include <chrono>
using namespace std::chrono;

#include <plog/Log.h>

#include "corecontroller.h"
#include "packet.h"
#include "config.hpp"


CoreController::CoreController(Core* parent, Router* router, Scheduler* scheduler, NeuronBlock* neuron_block, std::vector<CSRAMRow*> csram, std::vector<int> neuron_instructions) {
	this->parent = parent;
	this->router = router;
	this->scheduler = scheduler;
	this->neuron_block = neuron_block;
	this->csram = csram;

	this->neuron_instructions = neuron_instructions;
	
	for (int i = 0; i < NUM_NEURONS; i++) {
		csram_dereferenced[i] = *csram[i];
	}
	
	for (int i = 0; i < NUM_AXONS; i++) {
		neuron_instructions_dereferenced[i] = neuron_instructions[i];
	}

	this->routerPercentage = 0;
}

CoreController::~CoreController() {

}

std::string CoreController::getSpikes() {
	std::string str;
	std::stringstream sstream;
	char *pEnd = NULL;
	for (int i = 0; i < Config::parameters["num_axons"].GetInt(); i += 4) {
		str = "";
		for (int j = 0; j < 4; j++) {
			str += std::to_string(spikes[i+j]);
		}
		int temp = (int)std::strtol(str.c_str(), &pEnd, 2);
		sstream << std::hex << temp;
	}
	
	return sstream.str();
}	