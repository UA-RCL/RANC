#include "hip/hip_runtime.h"
/// RANC.cpp
/// 
/// Created for the University of Arizona Reconfigurable Computing Lab
///
///

#include <iostream>

#include <plog/Log.h>
#include <chrono>
using namespace std::chrono;

#include "config.hpp"
#include "rancgrid.h"
#include "router.h"
#include "scheduler.h"
#include "neuronblock.h"
#include "csramrow.h"
#include "corecontroller.h"
#include "schedulersram.h"

#define NEURONBLOCK_VERSION 2
#define BLOCK_SIZE 128

RANCGrid::RANCGrid(std::vector<std::vector<Packet>> input_packets, std::vector<Core*> components, OutputBus* output_bus) {
	this->components = components;
	this->input_packets = input_packets;
	this->output_bus = output_bus;
};

void print_array(int * a, int size);

__global__ void neuronBlockCore(CSRAMRow* csram, bool* srams, int* curr_word_index, int* neuron_instructions, bool* chip_output_spikes, int reset_type, int core, int sram_size, int num_axons, int num_neurons) {
	
	bool* input_spikes = srams + core*sram_size;
	input_spikes += curr_word_index[core]*num_axons;
	
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	if (i >= num_neurons) return;
	
	CSRAMRow* csram_row = &csram[core*num_neurons+i];
	int current_potential = csram_row->current_potential;

	// Get active connecitons (where there is both a spike and connection)
	for (int axon = 0; axon < num_axons; axon++) {
		if (csram_row->connections[axon] && input_spikes[axon]) {
			current_potential += csram_row->weights[neuron_instructions[core*num_axons+axon]];
		}
	}
	
	// Apply leak
	current_potential += csram_row->leak;


	// Check for spike
	if (current_potential >= csram_row->positive_threshold) {
		chip_output_spikes[core*num_neurons+i] = true;
	}
	else {
		chip_output_spikes[core*num_neurons+i] = false;
	}

	// Send potential back to csram
	int positive_threshold = csram_row->positive_threshold;
	int negative_threshold = csram_row->negative_threshold;
	int reset_potential = csram_row->reset_potential;
	int reset_mode = csram_row->reset_mode;
	
	switch(reset_type){
		case asymetric_reset:
			if( current_potential >= positive_threshold){
				switch (reset_mode){
					case absolute_reset:
						csram_row->current_potential =  reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - positive_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}
			} else if( current_potential < negative_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  -reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - negative_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out range of acceptable reset modes.");
						return;
				}
			} else {
				csram_row->current_potential =  current_potential;
				return;
			}
			break;
		//-----------------------------------------------------------//
		case symetric_reset:
			if( current_potential >= positive_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - positive_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}

			}else if( current_potential <= negative_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  -reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - negative_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}
			}else{
				csram_row->current_potential =  current_potential;
				return;
			}
			break;
		//-----------------------------------------------------------//
		default:
			printf( "[ERROR] Reset Type not accepted at this time.");
			return;
	}
		
	/*if (neuron_block_trace_verbosity == nb_trace_potentials) {
		LOG_DEBUG_(1) << "\tNeuron ends at potential: " << (*csram_row)->current_potential;
	}*/
	
	
}

__global__ void neuronBlockGrid(CSRAMRow* csram, bool* srams, int curr_word_index, int* neuron_instructions, bool* output_spikes, int reset_type, int chip_neuron_count, int num_axons, int num_neurons, int max_tick_offset) {
	// TODO: Shared and constant memory utilization
	int threadID = threadIdx.x+blockIdx.x*blockDim.x;
	if (threadID >= chip_neuron_count) return;

	CSRAMRow* csram_row = &csram[threadID];
	int current_potential = csram_row->current_potential;
	int neuron_instructions_offset = (threadID / num_neurons) * num_axons;
	int axons_offset = (threadID / num_neurons) * num_axons * max_tick_offset + curr_word_index * num_axons;

	// Get active connections (where there is both a spike and connection)
	for (int i = 0; i < num_axons; i++) {
		current_potential += (csram_row->weights[neuron_instructions[neuron_instructions_offset + i]] * csram_row->connections[i] * srams[axons_offset + i]);
	}

	// Apply leak
	current_potential += csram_row->leak;

	// Check for spike
	if (current_potential >= csram_row->positive_threshold) {
		output_spikes[threadID] = true;
	}
	else {
		output_spikes[threadID] = false;
	}

	// Send potential back to csram
	int positive_threshold = csram_row->positive_threshold;
	int negative_threshold = csram_row->negative_threshold;
	int reset_potential = csram_row->reset_potential;
	int reset_mode = csram_row->reset_mode;

	switch(reset_type){
		case asymetric_reset:
			if( current_potential >= positive_threshold){
				switch (reset_mode){
					case absolute_reset:
						csram_row->current_potential =  reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - positive_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}
			} else if( current_potential < negative_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  -reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - negative_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out range of acceptable reset modes.");
						return;
				}
			} else {
				csram_row->current_potential =  current_potential;
				return;
			}
			break;
		//-----------------------------------------------------------//
		case symetric_reset:
			if( current_potential >= positive_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - positive_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}

			}else if( current_potential <= negative_threshold){
				switch(reset_mode){
					case absolute_reset:
						csram_row->current_potential =  -reset_potential;
						return;
					case linear_reset:
						csram_row->current_potential =  current_potential - negative_threshold;
						return;
					default:
						printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
						return;
				}
			}else{
				csram_row->current_potential =  current_potential;
				return;
			}
			break;
		//-----------------------------------------------------------//
		default:
			printf( "[ERROR] Reset Type not accepted at this time.");
			return;
	}

}

__global__ void neuronBlockSynpse(CSRAMRow* csram, bool* srams, int curr_word_index, int* neuron_instructions, bool* output_spikes, int reset_type, int chip_neuron_count, int num_axons, int num_neurons, int max_tick_offset) {
	// TODO: Shared and constant memory utilization for input_spikes
	int threadID = threadIdx.x+blockIdx.x*blockDim.x;
	if (threadID >= chip_neuron_count*num_axons/2) return;

	int axons_offset = blockIdx.x/num_neurons * num_axons * max_tick_offset + curr_word_index * num_axons;
	
	CSRAMRow* csram_row = &csram[blockIdx.x];
	extern __shared__ int current_potential[];	

	/********************** Shared Memory phase **********************/
    int tmp1 = int(csram_row->connections[threadIdx.x]) * int(srams[axons_offset + threadIdx.x]);
    int tmp2 = int(csram_row->connections[threadIdx.x+blockDim.x]) * int(srams[axons_offset + threadIdx.x + blockDim.x]);
    current_potential[threadIdx.x] = tmp1 * csram_row->weights[neuron_instructions[((blockIdx.x/num_neurons) * num_axons) + threadIdx.x]]
                                    + tmp2 * csram_row->weights[neuron_instructions[((blockIdx.x/num_neurons) * num_axons) + threadIdx.x + blockDim.x]];
    __syncthreads();

	/********************** Reduction Tree ****************************/
    for(unsigned int stride = blockDim.x/ 2; stride > 0; stride >>= 1){
        if(threadIdx.x < stride){
            current_potential[threadIdx.x] = current_potential[threadIdx.x] + current_potential[threadIdx.x+stride];
        }
        __syncthreads();
    }

	/*********************** Neuron update *****************************/
    if (threadIdx.x == 0) {
        int current_potential_local;
        current_potential_local = current_potential[0] + csram_row->current_potential;

        // Apply leak
        current_potential_local += csram_row->leak;

        // Check for spike
        if (current_potential_local >= csram_row->positive_threshold) {
            output_spikes[blockIdx.x] = true;
            //router->receiveLocal(Packet((*csram_row)->dx, (*csram_row)->dy, (*csram_row)->destination_tick, (*csram_row)->destination_axon));
        } else {
            output_spikes[blockIdx.x] = false;
        }

        // Send potential back to csram
        int positive_threshold = csram_row->positive_threshold;
        int negative_threshold = csram_row->negative_threshold;
        int reset_potential = csram_row->reset_potential;
        int reset_mode = csram_row->reset_mode;

        switch (reset_type) {
            case asymetric_reset:
                if (current_potential_local >= positive_threshold) {
                    switch (reset_mode) {
                        case absolute_reset:
                            csram_row->current_potential = reset_potential;
                            return;
                        case linear_reset:
                            csram_row->current_potential = current_potential_local - positive_threshold;
                            return;
                        default:
                            printf("[ERROR] Reset mode out of range of acceptable reset modes.");
                            return;
                    }
                } else if (current_potential_local < negative_threshold) {
                    switch (reset_mode) {
                        case absolute_reset:
                            csram_row->current_potential = -reset_potential;
                            return;
                        case linear_reset:
                            csram_row->current_potential = current_potential_local - negative_threshold;
                            return;
                        default:
                            printf("[ERROR] Reset mode out range of acceptable reset modes.");
                            return;
                    }
                } else {
                    csram_row->current_potential = current_potential_local;
                    return;
                }
                break;
                //-----------------------------------------------------------//
            case symetric_reset:
                if (current_potential_local >= positive_threshold) {
                    switch (reset_mode) {
                        case absolute_reset:
                            csram_row->current_potential = reset_potential;
                            return;
                        case linear_reset:
                            csram_row->current_potential = current_potential_local - positive_threshold;
                            return;
                        default:
                            printf("[ERROR] Reset mode out of range of acceptable reset modes.");
                            return;
                    }

                } else if (current_potential_local <= negative_threshold) {
                    switch (reset_mode) {
                        case absolute_reset:
                            csram_row->current_potential = -reset_potential;
                            return;
                        case linear_reset:
                            csram_row->current_potential = current_potential_local - negative_threshold;
                            return;
                        default:
                            printf("[ERROR] Reset mode out of range of acceptable reset modes.");
                            return;
                    }
                } else {
                    csram_row->current_potential = current_potential_local;
                    return;
                }
                break;
                //-----------------------------------------------------------//
            default:
                printf("[ERROR] Reset Type not accepted at this time.");
                return;
        }

        /*if (neuron_block_trace_verbosity == nb_trace_potentials) {
            LOG_DEBUG_(1) << "\tNeuron ends at potential: " << (*csram_row)->current_potential;
        }*/

    }

}

__global__ void schedulerKernel(bool* srams, int* curr_word_indexes, int max_tick_offset, int num_axons, int num_cores) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int core = i / num_axons;
	int axon = i % num_axons;

	int core_starting_index = core*(num_axons*max_tick_offset);
	
	srams[core_starting_index + curr_word_indexes[core]*num_axons + axon] = 0;
	
	if (i < num_cores) {
		if (curr_word_indexes[i] == max_tick_offset - 1) {
			curr_word_indexes[i] = 0;
		} else {
			curr_word_indexes[i]++;
		}
	}
	
}

__global__ void packetKernel(Packet* packets, int num_packets, bool* srams, int* curr_word_indexes, int num_axons, int num_neurons, int max_tick_offset, int width_cores, int num_cores) {
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i >= num_packets) return;
	
	int coreNum = packets[i].dx + packets[i].dy * width_cores;
		
	if (coreNum >= num_cores) {
		printf("[ERROR] Input packet tried to leave the grid to core %d (there are %d cores)\n", coreNum, gridDim.x*blockDim.x/num_neurons);
	}
	
	int word = packets[i].delivery_tick + 1;
	int axon = packets[i].destination_axon;
	
	// Increment word so that it is not written to current timestep

	// Offset by current word
	if (word + curr_word_indexes[coreNum] >= max_tick_offset) {
		word += curr_word_indexes[coreNum] - max_tick_offset;
	} else {
		word += curr_word_indexes[coreNum];
	}
	
	if (word == curr_word_indexes[coreNum]) {
		printf("[WARNING] Packet tried to write to current word in scheduler (core (%d, %d), word %d)", coreNum % width_cores, coreNum / width_cores, word);
	} else if (srams[coreNum*(num_axons*max_tick_offset)+word*num_axons+axon] == 1) {
		printf("[WARNING] Scheduler received duplicate external spike in same time tick (core (%d, %d), word %d)", coreNum % width_cores, coreNum / width_cores, word);
	} else {
		srams[coreNum*(num_axons*max_tick_offset)+word*num_axons+axon] = 1;
	}
	
}

__global__ void routerKernel(CSRAMRow* csrams, bool* chip_output_spikes, bool* srams, int* curr_word_indexes, int num_axons, int num_neurons, int max_tick_offset, int width_cores, int num_cores) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int core = i / num_neurons;
	int neuron = i % num_neurons;
	
	if (!chip_output_spikes[core*num_neurons+neuron]) return;
		
	int coreNum = core + csrams[core*num_neurons+neuron].dx + csrams[core*num_neurons+neuron].dy * width_cores;
	
	if (coreNum >= num_cores) {
		printf("[ERROR] Generated packet tried to leave the grid\n");
	}
	
	int word = csrams[core*num_neurons+neuron].destination_tick + 1;
	int axon = csrams[core*num_neurons+neuron].destination_axon;
	
	// Increment word so that it is not written to current timestep

	// Offset by current word
	if (word + curr_word_indexes[coreNum] >= max_tick_offset) {
		word += curr_word_indexes[coreNum] - max_tick_offset;
	} else {
		word += curr_word_indexes[coreNum];
	}
	
	if (word == curr_word_indexes[coreNum]) {
		printf("[WARNING] Packet tried to write to current word in scheduler (core (%d, %d), word %d)\n", coreNum % width_cores, coreNum / width_cores, word);
	} else if (srams[coreNum*(num_axons*max_tick_offset)+word*num_axons+axon] == 1) {
		printf("[WARNING] Scheduler received duplicate internal spike in same time tick (core (%d, %d), word %d)\n", coreNum % width_cores, coreNum / width_cores, word);
	} else {
		srams[coreNum*(num_axons*max_tick_offset)+word*num_axons+axon] = 1;
	}
	
}

void dump_sram(bool* srams_device, bool* srams, int num_cores, int num_axons, int max_tick_offset, int curr_word_index){
	hipError_t err = hipMemcpy(srams, srams_device, num_cores*num_axons*max_tick_offset*sizeof(bool), hipMemcpyDeviceToHost);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	
	for (int i = 0; i < num_cores; i++) {
		for (int j = 0; j < num_axons; j++) {
			std::cout << srams[i*num_axons*max_tick_offset+curr_word_index*num_axons+j];
		}
		std::cout << std::endl;
	}
}

void print_array(int * a, int size){
	printf("\n##############################Array##############################\n");
	for(int i = 0; i < size; i++){
	printf("%d ", a[i]);
	}
	printf("\n##############################End of Array##############################\n\n");
	return;
}

void RANCGrid::beginActivity(int num_ticks, int report_frequency) {
	
	long int schedulerTime = 0;
	long int neuronBlockTime = 0;
	long int routerTime = 0;
	long int outputTime = 0;
	long int GPULoadTime = 0;
	
	hipError_t err;
	int gridDimension;
	int blockDimension;

	// save some parameters for easier use
	int num_cores = Config::parameters["num_cores_x"].GetInt()*Config::parameters["num_cores_y"].GetInt();
	int num_neurons = Config::parameters["num_neurons"].GetInt();
	int num_axons = Config::parameters["num_axons"].GetInt();
	int total_neurons = num_cores*num_neurons;
	int max_tick_offset = Config::parameters["max_tick_offset"].GetInt();
	int curr_word_index = Config::parameters["max_tick_offset"].GetInt() - 1;;
	int output_core = output_bus->x + output_bus->y*Config::parameters["num_cores_x"].GetInt();
	int sram_size = sizeof(bool)*Config::parameters["max_tick_offset"].GetInt()*Config::parameters["num_axons"].GetInt();
	
	std::cout << "Input file parsing done, now initializing GPU memory." << std::endl;
	
	auto clock = high_resolution_clock::now();
	
	// BEGIN ----- Scheduler, Router, Packet variables -----
	
	// sram CPU copy
	bool* srams = (bool*)malloc(num_cores*sram_size);
	if (srams == NULL) std::cout << "[ERROR] Memory allocation for SRAM failed, simulation will fail";
	
	// sram GPU copy
	bool* srams_device;
	err = hipMalloc((void **) &srams_device, num_cores*sram_size);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

	// curr_word_indexes CPU&GPU copy
	int* curr_word_indexes = (int*)malloc(num_cores*sizeof(int));
	int* curr_word_indexes_device;
	err = hipMalloc((void **) &curr_word_indexes_device, num_cores*sizeof(int));
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	
	// populate CPU sram and curr_word_indexes data from serialized data
	for (int core = 0; core < num_cores; core++) {
		for (int tick = 0; tick < Config::parameters["max_tick_offset"].GetInt(); tick++) {
			for (int axon = 0; axon < Config::parameters["num_axons"].GetInt(); axon++) {
				srams[core*sram_size + tick*Config::parameters["num_axons"].GetInt() + axon] = components[core]->scheduler->sram->data[tick*Config::parameters["num_axons"].GetInt()+axon];
			}
		}
		curr_word_indexes[core] = components[core]->scheduler->sram->curr_word_index;
	}
	
	// copy sram and curr_word_indexes data to GPU
	err = hipMemcpy(srams_device, srams, num_cores*sram_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	err = hipMemcpy(curr_word_indexes_device, curr_word_indexes, num_cores*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	
	Packet* packets_device;
	
	
	// END ----- Scheduler, Router, Packet variables -----
	
	
	
	
	// BEGIN ----- NeuronBlock variables -----
	
	// Create CSRAM GPU data copy
	int ranc_num_cores = Config::parameters["num_cores_x"].GetInt() * Config::parameters["num_cores_y"].GetInt();
	CSRAMRow *chip_csram_host;
	CSRAMRow *chip_csram_device;
    int *chip_neuron_instructions_host;
    int *chip_neuron_instructions_device;
    chip_csram_host = (CSRAMRow *)calloc(ranc_num_cores * num_neurons, sizeof(CSRAMRow));
    chip_neuron_instructions_host = (int *)calloc(ranc_num_cores * NUM_AXONS, sizeof(int));

    if (hipMalloc((void**) &chip_csram_device, ranc_num_cores * num_neurons * sizeof(CSRAMRow)) != hipSuccess){
        printf("Failed to allocate memory for Chip level CSRAM on GPU\n");
    }
    if (hipMalloc((void**) &chip_neuron_instructions_device, ranc_num_cores * NUM_AXONS * sizeof(int)) != hipSuccess){
        printf("Failed to allocate memory for Chip level Neuron Instructions on GPU\n");
    }

    // Copy values from cores into chip level csram and neuron instructions
    int neuron_offset = 0;
    int axon_offset = 0;
    for(auto core: components){
        for (int i=0; i < num_neurons; i++){
            chip_csram_host[neuron_offset + i] = core->core_controller->csram_dereferenced[i];
        }
        neuron_offset += num_neurons;

        for (int i = 0; i < NUM_AXONS; i++){
            chip_neuron_instructions_host[axon_offset + i] = core->core_controller->neuron_instructions_dereferenced[i];
        }
        axon_offset += NUM_AXONS;
    }
    // Copy values to device
    if (hipMemcpy(chip_csram_device, chip_csram_host, ranc_num_cores * num_neurons * sizeof(CSRAMRow), hipMemcpyHostToDevice) != hipSuccess){
        printf("Failed to copy Chip level CSRAM to device\n");
    }
    if (hipMemcpy(chip_neuron_instructions_device, chip_neuron_instructions_host, ranc_num_cores * NUM_AXONS * sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
        printf("Failed to copy Chip level Neuron Instructions to device\n");
    }

    // Allocate chip level output spike array
    bool *chip_output_spikes_device;
    bool *chip_output_spikes_host = (bool *)calloc(ranc_num_cores * num_neurons, sizeof(bool));
    if (hipMalloc((void **)&chip_output_spikes_device, ranc_num_cores * num_neurons * sizeof(bool)) != hipSuccess){
        printf("Failed to allocate device memory for chip level output spikes\n");
    }
	
	
	// END ----- NeuronBlock variables -----
	
	GPULoadTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
	std::cout << "Done allocating and copying GPU memory, cumulative overhead of " << GPULoadTime << " microseconds." << std::endl;
	
	
	std::cout << "Starting simulation with " << num_ticks << " ticks." << std::endl;
	
	if (Config::traceSpecified()) {
		LOG_DEBUG_(1) << "Starting simulation with " << num_ticks << " ticks.";
	}

	// FIXME: max_tick_offset of 16 will actually correspond to a max of 15 ticks in the future being able to be specified. We need error checking for this as well.	
	if (input_packets.size() > 0) {
		int numPackets = input_packets[0].size();
		if (numPackets != 0) {
			gridDimension = (numPackets + BLOCK_SIZE - 1) / BLOCK_SIZE;
			blockDimension = min(numPackets, BLOCK_SIZE);
			
			err = hipMalloc((void **) &packets_device, numPackets*sizeof(Packet));
			if (err != hipSuccess) printf("Error in prerouting: %s\n", hipGetErrorString(err));
			
			err = hipMemcpy(packets_device, &(input_packets[0][0]), sizeof(Packet)*(numPackets), hipMemcpyHostToDevice);
			if (err != hipSuccess) printf("Error in prerouting: %s\n", hipGetErrorString(err));
			
			packetKernel<<<gridDimension, blockDimension>>>(packets_device, numPackets, srams_device, curr_word_indexes_device, num_axons, num_neurons, max_tick_offset, Config::parameters["num_cores_x"].GetInt(), num_cores);
			err = hipGetLastError();
			if (err != hipSuccess) printf("Error in prerouting: %s\n", hipGetErrorString(err));
			err = hipDeviceSynchronize();
			if (err != hipSuccess) printf("Error in prerouting: %s\n", hipGetErrorString(err));
		}
	}
	
	// Iterate through each tick
	for (unsigned int tick = 1; tick <= num_ticks; tick++) {		
		
		if (tick % report_frequency == 0) {
			std::cout << "Tick " << tick << " started" << std::endl;
		}

		if (Config::traceSpecified()) {
			LOG_DEBUG_(1) << "-------------------- Tick " << tick << " begins --------------------";
		}
		
		// KERNEL BEGIN: Scheduler
		clock = high_resolution_clock::now();
		
		int total_axons = num_cores * num_axons;
		gridDimension = (total_axons + BLOCK_SIZE - 1) / BLOCK_SIZE;
		blockDimension = min(total_axons, BLOCK_SIZE);
		
		schedulerKernel<<<gridDimension, blockDimension>>>(srams_device, curr_word_indexes_device, max_tick_offset, num_axons, num_cores);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in scheduler: %s\n", hipGetErrorString(err));
		
		err = hipDeviceSynchronize();
		if (err != hipSuccess) printf("Error in scheduler: %s\n", hipGetErrorString(err));
			
		schedulerTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
		// KERNEL END: Scheduler
		
		
		
		
		
		
		if (curr_word_index == Config::parameters["max_tick_offset"].GetInt() - 1) {
			curr_word_index = 0;
		} else {
			curr_word_index++;
		}
		
		
		
		// KERNEL BEGIN: Packet
		clock = high_resolution_clock::now();
		
		if (input_packets.size() > tick) {

			int numPackets = input_packets[tick].size();
			if (numPackets != 0) {
				gridDimension = (numPackets + BLOCK_SIZE - 1) / BLOCK_SIZE;
				blockDimension = min(numPackets, BLOCK_SIZE);
				
				err = hipMalloc((void **) &packets_device, numPackets*sizeof(Packet));
				if (err != hipSuccess) printf("Error in packet: %s\n", hipGetErrorString(err));
				
				err = hipMemcpy(packets_device, &(input_packets[tick][0]), sizeof(Packet)*(numPackets), hipMemcpyHostToDevice);
				if (err != hipSuccess) printf("Error in packet: %s\n", hipGetErrorString(err));
				
				packetKernel<<<gridDimension, blockDimension>>>(packets_device, numPackets, srams_device, curr_word_indexes_device, num_axons, num_neurons, max_tick_offset, Config::parameters["num_cores_x"].GetInt(), num_cores);
				err = hipGetLastError();
				if (err != hipSuccess) printf("Error in packet: %s\n", hipGetErrorString(err));
				
				err = hipDeviceSynchronize();
				if (err != hipSuccess) printf("Error in packet: %s\n", hipGetErrorString(err));
		
				hipFree(packets_device);
			}

		}
	
		routerTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
		// KERNEL END: Packet
		
		
		
		// KERNEL BEGIN: NeuronBlock
		clock = high_resolution_clock::now();
		
		if (hipMemcpy(chip_output_spikes_device, chip_output_spikes_host, ranc_num_cores * num_neurons * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess){
			printf("Failed to copy device memory from chip level output spikes\n");
		}
		/*if (hipMemset(chip_output_spikes_device, 0, ranc_num_cores * num_neurons * sizeof(bool)) != hipSuccess){
			printf("Failed to wipe device chip level output spikes\n");
		}*/
		
		
		// Original core-level parallelism kernel (thread->neuron)
		if (NEURONBLOCK_VERSION == 0) {
			for (int core = 0; core < num_cores; core++) {
			
				gridDimension = (num_neurons + BLOCK_SIZE - 1) / BLOCK_SIZE;
				blockDimension = min(num_neurons, BLOCK_SIZE);
								
				neuronBlockCore<<<gridDimension, blockDimension>>>(chip_csram_device,
																srams_device,
																curr_word_indexes_device,
																chip_neuron_instructions_device,
																chip_output_spikes_device,
																Config::parameters["neuron_reset_type"].GetInt(),
																core,
																sram_size,
																num_axons,
																num_neurons);
				err = hipGetLastError();
				if (err != hipSuccess) printf("Error in neuronBlock: %s\n", hipGetErrorString(err));
				
			}

		}
		
		// Sahil's grid-level parallelism kernel (thread->neuron)
		else if (NEURONBLOCK_VERSION == 1) {

			gridDimension = ceil(total_neurons/(float)blockDimension);
			blockDimension = min(total_neurons, BLOCK_SIZE);
			
			neuronBlockGrid<<<gridDimension, blockDimension>>>(chip_csram_device,
														   srams_device,
														   curr_word_index,
														   chip_neuron_instructions_device,
														   chip_output_spikes_device,
														   Config::parameters["neuron_reset_type"].GetInt(),
														   ranc_num_cores*num_axons,
														   num_axons,
														   num_neurons,
														   max_tick_offset);
			err = hipGetLastError();
			if (err != hipSuccess) printf("[cudaKernelLaunch]: %s\n", hipGetErrorString(err));
			
		}
		
		//Ilkin's grid-synapse-level kernel (thread->axon)
		else if (NEURONBLOCK_VERSION == 2) {
			
			blockDimension = num_axons/2;
			gridDimension = ranc_num_cores * num_neurons;
			
			neuronBlockSynpse<<<gridDimension, blockDimension, sizeof(int)*num_axons/2>>>(chip_csram_device,
														   srams_device,
														   curr_word_index,
														   chip_neuron_instructions_device,
														   chip_output_spikes_device,
														   Config::parameters["neuron_reset_type"].GetInt(),
														   ranc_num_cores*num_axons,
														   num_axons,
														   num_neurons,
														   max_tick_offset);
			err = hipGetLastError();
			if (err != hipSuccess) printf("[cudaKernelLaunch]: %s\n", hipGetErrorString(err));
		}
		
		else {
			std::cout << "ERROR: Invalid neuronblock kernel version selected. Simulation will fail." << std::endl;
			return;
		}
		
		err = hipDeviceSynchronize();
		if (err != hipSuccess) printf("Error in neuronBlock: %s\n", hipGetErrorString(err));
		
		// Need to copy back the output core's spikes, regardless of neruonBlock version
		bool* output_core_spikes_device = srams_device + output_core*sram_size + curr_word_index*num_axons;

		err = hipMemcpy(&components[output_core]->scheduler->sram->data[curr_word_index*num_axons], output_core_spikes_device, num_axons*sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess) printf("Error in neuronblock/output copyback: %s\n", hipGetErrorString(err));
		

		neuronBlockTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
		// KERNEL END: NeuronBlock
		
		
		
		
		// SERIAL BEGIN: write output_bus output spikes to output
		clock = high_resolution_clock::now();
		
		output_bus->scheduler->sram->curr_word_index = curr_word_index;
		LOG_INFO_(0) << output_bus->scheduler->printCurrentSpikes();
		
		outputTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
		// SERIAL END: end write output_bus output spikes to output
		
		
		

		// KERNEL BEGIN: Router
		clock = high_resolution_clock::now();
		
		int numCoresX = Config::parameters["num_cores_x"].GetInt(); //Equivalent to Width
		gridDimension = (total_neurons + BLOCK_SIZE - 1) / BLOCK_SIZE;
		blockDimension = min(total_neurons, BLOCK_SIZE);
		
		routerKernel<<<gridDimension, blockDimension>>>(chip_csram_device, chip_output_spikes_device, srams_device, curr_word_indexes_device, num_axons, num_neurons, max_tick_offset, numCoresX, num_cores);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in router execution: %s\n", hipGetErrorString(err));

		err = hipDeviceSynchronize();
		if (err != hipSuccess) printf("Error in router synchronize: %s\n", hipGetErrorString(err));
			
		routerTime += duration_cast<microseconds>(high_resolution_clock::now() - clock).count();
		// KERNEL BEGIN: Router
	}
	
	std::cout << "cumulative scheduler operations took " << schedulerTime << " microseconds" << std::endl;
	std::cout << "cumulative neuronBlock operations took " << neuronBlockTime << " microseconds" << std::endl;
	std::cout << "cumulative router operations took " << routerTime << " microseconds" << std::endl;
	std::cout << "cumulative output operations took " << outputTime << " microseconds" << std::endl;
	
	// FIXME: Please free the cuda memories
	hipFree(srams_device);
	hipFree(curr_word_indexes_device);
	
	hipFree(chip_csram_device);
	hipFree(chip_neuron_instructions_device);
	hipFree(chip_output_spikes_device);
	
}