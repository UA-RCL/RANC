/// neuronblock.cpp
/// 
/// Created for the University of Arizona Reconfigurable Computing Lab
///
///

#include<hip/hip_runtime.h>
#include <stdio.h>

#include "neuronblock.h"


__device__ __host__ NeuronBlock::NeuronBlock(){
    this->current_potential = 0;
    this->reset_type = 1;
}

__device__ __host__ NeuronBlock::NeuronBlock(int reset_type){
    this->current_potential = 0;
    this->reset_type = reset_type;
}

__device__ __host__ NeuronBlock::NeuronBlock(int reset_type, int current_potential){
    this->current_potential = current_potential;
    this->reset_type = reset_type;
}

__device__ __host__ void NeuronBlock::integrate(int synaptic_weights[NUM_WEIGHTS], int neuron_instruction) {
	current_potential += synaptic_weights[neuron_instruction];
}

__device__ __host__ void NeuronBlock::leak(int leak){
	current_potential += leak;
}

__device__ __host__ bool NeuronBlock::spikes(int positive_threshold){
	return current_potential >= positive_threshold;
}

__device__ __host__ int NeuronBlock::output_potential(int positive_threshold, int negative_threshold, int reset_potential, int reset_mode) {
	
	switch(reset_type){
            case asymetric_reset:
                if( current_potential >= positive_threshold){
                    switch (reset_mode){
                        case absolute_reset:
                            return reset_potential;
                        
                        case linear_reset:
                            return current_potential - positive_threshold;

                        default:
                            printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
                    }
                } else if( current_potential < negative_threshold){
                    switch(reset_mode){
                        case absolute_reset:
                            return -reset_potential;

                        case linear_reset:
                            return current_potential - negative_threshold;

                        default:
                            printf( "[ERROR] Reset mode out range of acceptable reset modes.");
                    }
                } else {
                    return current_potential;
                }
                break;
            //-----------------------------------------------------------//
            case symetric_reset:
                if( current_potential >= positive_threshold){
                    switch(reset_mode){
                        case absolute_reset:
                            return reset_potential;

                        case linear_reset:
                            return current_potential - positive_threshold;

                        default:
                            printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
                    }

                }else if( current_potential <= negative_threshold){
                    switch(reset_mode){
                        case absolute_reset:
                            return -reset_potential;

                        case linear_reset:
                            return current_potential - negative_threshold;

                        default:
                            printf( "[ERROR] Reset mode out of range of acceptable reset modes.");
                    }
                }else{
                    return current_potential;
                }
                break;
            //-----------------------------------------------------------//
            default:
                printf( "[ERROR] Reset Type not accepted at this time.");
        }
	printf( "[ERROR] Reset Type not accepted at this time.");
	return 0;
}
